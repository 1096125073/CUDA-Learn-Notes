#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <vector>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>
#include <hip/hip_fp8.h>
#include <torch/types.h>
#include <torch/extension.h>

#define WARP_SIZE 32
#define INT4(value) (reinterpret_cast<int4*>(&(value))[0])
#define FLOAT4(value) (reinterpret_cast<float4*>(&(value))[0])

// -------------------------------------- FP32 -------------------------------------- 
// Warp Reduce Sum
template<const int kWarpSize = WARP_SIZE>
__device__ __forceinline__ float warp_reduce_sum_f32(float val) {
  #pragma unroll
  for (int mask = kWarpSize >> 1; mask >= 1; mask >>= 1) {
    val += __shfl_xor_sync(0xffffffff, val, mask);
  }
  return val;
}

// Block reduce sum/max/min device helper for Layer/RMS Norm/Softmax etc.
// grid 1D block 1D, grid(N/256), block(256)
template<const int NUM_THREADS=256>
__device__ __forceinline__ float block_reduce_sum_f32(float val) {
  // always <= 32 warps per block (limited by 1024 threads per block)
  constexpr int NUM_WARPS = (NUM_THREADS + WARP_SIZE - 1) / WARP_SIZE;
  int warp = threadIdx.x / WARP_SIZE;
  int lane = threadIdx.x % WARP_SIZE;
  static __shared__ float shared[NUM_WARPS];
  
  val = warp_reduce_sum_f32<WARP_SIZE>(val);
  if (lane == 0) shared[warp] = val;
  __syncthreads();
  val = (lane < NUM_WARPS) ? shared[lane] : 0.0f;
  val = warp_reduce_sum_f32<NUM_WARPS>(val);
  return val;
}

// RMS Norm: x: NxK(K=256<1024), y': NxK, y'=x/rms(x) each row
// 1/rms(x) = rsqrtf( sum(x^2)/K ) each row
// grid(N*K/K), block(K<1024) N=batch_size*seq_len, K=hidden_size
// y=y'*g (g: scale)
template<const int NUM_THREADS=256>
__global__ void rms_norm_f32(float* x, float* y, float g, int N, int K) {
  int tid = threadIdx.x; // 0..K-1
  int bid = blockIdx.x; // 0..N-1
  int idx = bid * blockDim.x + threadIdx.x;
  const float epsilon = 1e-5f;

  __shared__ float s_variance; // shared within block
  float value = (idx < N * K) ? x[idx] : 0.0f; // load once only
  float variance = value * value;
  variance = block_reduce_sum_f32<NUM_THREADS>(variance);
  if (tid == 0) s_variance = rsqrtf(variance / (float) K + epsilon);
  // wait for s_variance in shared memory to be ready for all threads
  __syncthreads(); 
  if (idx < N * K) y[idx] = (value * s_variance) * g;
}

// RMS Norm Vec4: x: NxK(K=256<1024), y': NxK, y'=x/rms(x) each row
// 1/rms(x) = rsqrtf( sum(x^2)/K ) each row
// grid(N*K/K), block(K/4<1024) N=batch_size*seq_len, K=hidden_size
// y=y'*g (g: scale)
template<const int NUM_THREADS=256/4>
__global__ void rms_norm_f32x4(float* x, float* y, float g, int N, int K) {
  int tid = threadIdx.x; // 0..K-1
  int bid = blockIdx.x; // 0..N-1
  int idx = (bid * blockDim.x + threadIdx.x) * 4;
  const float epsilon = 1e-5f;

  __shared__ float s_variance; // shared within block
  float4 reg_x = FLOAT4(x[idx]);
  float variance = (idx < N * K) ? (reg_x.x * reg_x.x + reg_x.y * reg_x.y 
                                  + reg_x.z * reg_x.z + reg_x.w * reg_x.w) : 0.0f;
  variance = block_reduce_sum_f32<NUM_THREADS>(variance);
  if (tid == 0) s_variance = rsqrtf(variance / (float) K + epsilon);
  // wait for s_variance in shared memory to be ready for all threads
  __syncthreads(); 
  float4 reg_y;
  reg_y.x = reg_x.x * s_variance * g;
  reg_y.y = reg_x.y * s_variance * g;
  reg_y.z = reg_x.z * s_variance * g;
  reg_y.w = reg_x.w * s_variance * g;
  if (idx < N * K) FLOAT4(y[idx]) = reg_y;
}
