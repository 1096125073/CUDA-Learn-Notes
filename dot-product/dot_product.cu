#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <vector>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>
#include <hip/hip_fp8.h>
#include <torch/types.h>

#define WARP_SIZE 32
#define INT4(value) (reinterpret_cast<int4*>(&(value))[0])
#define FLOAT4(value) (reinterpret_cast<float4*>(&(value))[0])

// -------------------------------------- FP32 -------------------------------------- 
// Warp Reduce Sum
template<const int kWarpSize = WARP_SIZE>
__device__ __forceinline__ float warp_reduce_sum_f32(float val) {
  #pragma unroll
  for (int mask = kWarpSize >> 1; mask >= 1; mask >>= 1) {
    val += __shfl_xor_sync(0xffffffff, val, mask);
  }
  return val;
}

// Dot Product
// grid(N/256), block(256)
// a: Nx1, b: Nx1, y=sum(elementwise_mul(a,b))
template<const int NUM_THREADS = 256>
__global__ void dot_prod_f32_acc_with_f32_kernel(
  float* a, float* b, float* y, int N) {
  int tid = threadIdx.x;
  int idx = blockIdx.x * NUM_THREADS + tid;
  constexpr int NUM_WARPS = (NUM_THREADS + WARP_SIZE - 1) / WARP_SIZE;
  __shared__ float reduce_smem[NUM_WARPS];

  // keep the data in register is enougth for warp operaion.
  float prod = (idx < N) ? a[idx] * b[idx] : 0.0f;
  int warp = tid / WARP_SIZE;
  int lane = tid % WARP_SIZE;
  // perform warp sync reduce.
  prod = warp_reduce_sum_f32<WARP_SIZE>(prod);
  // warp leaders store the data to shared memory.
  if (lane == 0) reduce_smem[warp] = prod;
  __syncthreads(); // make sure the data is in shared memory.
  // the first warp compute the final sum.
  prod = (lane < NUM_WARPS) ? reduce_smem[lane] : 0.0f;
  if (warp == 0) prod = warp_reduce_sum_f32<NUM_WARPS>(prod);
  if (tid == 0) atomicAdd(y, prod);
}

// Dot Product + Vec4
// grid(N/256), block(256/4)
// a: Nx1, b: Nx1, y=sum(elementwise_mul(a,b))
template<const int NUM_THREADS = 256/4>
__global__ void dot_prod_f32x4_acc_with_f32_kernel(
  float* a, float* b, float* y, int N) {
  int tid = threadIdx.x;
  int idx = (blockIdx.x * NUM_THREADS + tid) * 4;
  constexpr int NUM_WARPS = (NUM_THREADS + WARP_SIZE - 1) / WARP_SIZE;
  __shared__ float reduce_smem[NUM_WARPS];

  float4 reg_a = FLOAT4(a[idx]);
  float4 reg_b = FLOAT4(b[idx]);
  float prod = (idx < N) ? (reg_a.x * reg_b.x + reg_a.y * reg_b.y 
                          + reg_a.z * reg_b.z + reg_a.w * reg_b.w) : 0.0f;
  int warp = tid / WARP_SIZE;
  int lane = tid % WARP_SIZE;
  // perform warp sync reduce.
  prod = warp_reduce_sum_f32<WARP_SIZE>(prod);
  // warp leaders store the data to shared memory.
  if (lane == 0) reduce_smem[warp] = prod;
  __syncthreads(); // make sure the data is in shared memory.
  // the first warp compute the final sum.
  prod = (lane < NUM_WARPS) ? reduce_smem[lane] : 0.0f;
  if (warp == 0) prod = warp_reduce_sum_f32<NUM_WARPS>(prod);
  if (tid == 0) atomicAdd(y, prod);
}
