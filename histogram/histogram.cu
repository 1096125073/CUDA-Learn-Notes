#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <vector>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>
#include <hip/hip_fp8.h>
#include <torch/types.h>
#include <torch/extension.h>

#define WARP_SIZE 32
#define INT4(value) (reinterpret_cast<int4*>(&(value))[0])
#define FLOAT4(value) (reinterpret_cast<float4*>(&(value))[0])

// Histogram
// grid(N/256), block(256)
// a: Nx1, y: count histogram
__global__ void histogram_i32(int* a, int* y, int N) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < N) atomicAdd(&(y[a[idx]]), 1);
}

// Histogram + Vec4
// grid(N/256), block(256/4)
// a: Nx1, y: count histogram
__global__ void histogram_i32x4(int* a, int* y, int N) {
  int idx = 4 * (blockIdx.x * blockDim.x + threadIdx.x);
  if (idx < N) {
    int4 reg_a = INT4(a[idx]);
    atomicAdd(&(y[reg_a.x]), 1);
    atomicAdd(&(y[reg_a.y]), 1);
    atomicAdd(&(y[reg_a.z]), 1);
    atomicAdd(&(y[reg_a.w]), 1);
  }
}
