#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <vector>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>
#include <hip/hip_fp8.h>
#include <torch/types.h>
#include <torch/extension.h>

#define WARP_SIZE 32
#define INT4(value) (reinterpret_cast<int4*>(&(value))[0])
#define FLOAT4(value) (reinterpret_cast<float4*>(&(value))[0])

// -------------------------------------- FP32 -------------------------------------- 
// Warp Reduce Sum
template<const int kWarpSize = WARP_SIZE>
__device__ __forceinline__ float warp_reduce_sum_f32(float val) {
  #pragma unroll
  for (int mask = kWarpSize >> 1; mask >= 1; mask >>= 1) {
    val += __shfl_xor_sync(0xffffffff, val, mask);
  }
  return val;
}

// Block reduce sum/max/min device helper for Layer/RMS Norm/Softmax etc.
// grid 1D block 1D, grid(N/256), block(256)
template<const int NUM_THREADS=256>
__device__ __forceinline__ float block_reduce_sum_f32(float val) {
  // always <= 32 warps per block (limited by 1024 threads per block)
  constexpr int NUM_WARPS = (NUM_THREADS + WARP_SIZE - 1) / WARP_SIZE;
  int warp = threadIdx.x / WARP_SIZE;
  int lane = threadIdx.x % WARP_SIZE;
  static __shared__ float shared[NUM_WARPS];
  
  val = warp_reduce_sum_f32<WARP_SIZE>(val);
  if (lane == 0) shared[warp] = val;
  __syncthreads();
  val = (lane < NUM_WARPS) ? shared[lane] : 0.0f;
  val = warp_reduce_sum_f32<NUM_WARPS>(val);
  return val;
}

// Softmax x: N, y: N
// grid(N/256), block(K=256)
template<const int NUM_THREADS = 256>
__global__ void softmax_f32(float* x, float* y, float* total, int N) {
  const int tid = threadIdx.x;
  const int idx = blockIdx.x * blockDim.x + tid; 
  
  float exp_val = (idx < N) ? expf(x[idx]) : 0.0f;
  float sum = block_reduce_sum_f32<NUM_THREADS>(exp_val);
  // get the total sum of all blocks.
  if (tid == 0) atomicAdd(total, sum);
  __threadfence(); // grid level memory fence
  // e^x_i/sum(e^x_0,...,e^x_n-1) 
  if (idx < N) y[idx] = exp_val / (*total); 
}

// Softmax Vec4 x: N, y: N
// grid(N/256), block(256/4)
template<const int NUM_THREADS = 256/4>
__global__ void softmax_f32x4(float* x, float* y, float* total, int N) {
  const int tid = threadIdx.x;
  const int idx = (blockIdx.x * blockDim.x + tid) * 4; 
  
  float4 reg_x = FLOAT4(x[idx]);
  float4 reg_exp;
  reg_exp.x = (idx < N) ? expf(reg_x.x) : 0.0f;
  reg_exp.y = (idx < N) ? expf(reg_x.y) : 0.0f;
  reg_exp.z = (idx < N) ? expf(reg_x.z) : 0.0f;
  reg_exp.w = (idx < N) ? expf(reg_x.w) : 0.0f;
  float exp_val = (reg_exp.x + reg_exp.y + reg_exp.z + reg_exp.w);
  float sum = block_reduce_sum_f32<NUM_THREADS>(exp_val);
  // get the total sum of all blocks.
  if (tid == 0) atomicAdd(total, sum);
  __threadfence(); // grid level memory fence
  // e^x_i/sum(e^x_0,...,e^x_n-1) 
  if (idx < N) {
    float4 reg_y;
    reg_y.x = reg_exp.x / (*total);
    reg_y.y = reg_exp.y / (*total);
    reg_y.z = reg_exp.z / (*total);
    reg_y.w = reg_exp.w / (*total);
    FLOAT4(y[idx]) = reg_y; 
  }
}

// TODO: support per-token w/o __threadfence