#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <vector>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>
#include <hip/hip_fp8.h>
#include <torch/types.h>
#include <torch/extension.h>

#define WARP_SIZE 32
#define INT4(value) (reinterpret_cast<int4*>(&(value))[0])
#define FLOAT4(value) (reinterpret_cast<float4*>(&(value))[0])

// ElementWise Add  
// grid(N/256), block(256)
// a: Nx1, b: Nx1, c: Nx1, c = elementwise_add(a, b)
__global__ void elementwise_add_f32(float* a, float* b, float* c, int N) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < N) c[idx] = a[idx] + b[idx];
}

// ElementWise Add + Vec4
// grid(N/256), block(256/4)
// a: Nx1, b: Nx1, c: Nx1, c = elementwise_add(a, b)
__global__ void elementwise_add_f32x4(float* a, float* b, float* c, int N) {
  int idx = 4 * (blockIdx.x * blockDim.x + threadIdx.x);
  if (idx < N) {
    float4 reg_a = FLOAT4(a[idx]);
    float4 reg_b = FLOAT4(b[idx]);
    float4 reg_c;
    reg_c.x = reg_a.x + reg_b.x;
    reg_c.y = reg_a.y + reg_b.y;
    reg_c.z = reg_a.z + reg_b.z;
    reg_c.w = reg_a.w + reg_b.w;
    FLOAT4(c[idx]) = reg_c;
  }
}
